#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ 
void applyBuffer(FSystem* System, FTube** Buffer)
{
	int x = blockIdx.x;
	int y = threadIdx.x;

	//Update Chemicals
	if(System->Tubes[x][y].SrcX + Buffer[x][y].SrcX <= (SIZE - 1) && System->Tubes[x][y].SrcX + Buffer[x][y].SrcX >= 0.0f)
	{
		System->Tubes[x][y].SrcX = System->Tubes[x][y].SrcX + Buffer[x][y].SrcX;
	}
	else
	{
		//printf("x: %d Y: %d %lf\n", x, y, System->Tubes[x][y].SrcX + Buffer[x][y].SrcX);
	}
	if(System->Tubes[x][y].SrcY + Buffer[x][y].SrcY <= (SIZE - 1) && System->Tubes[x][y].SrcY + Buffer[x][y].SrcY >= 0.0f)
	{
		System->Tubes[x][y].SrcY = System->Tubes[x][y].SrcY + Buffer[x][y].SrcY;
	}
	else
	{
		//printf("x: %d Y: %d %lf\n", x, y, System->Tubes[x][y].SrcY + Buffer[x][y].SrcY);
	}
	if(System->Tubes[x][y].DesX + Buffer[x][y].DesX <= (SIZE - 1) && System->Tubes[x][y].DesX + Buffer[x][y].DesX >= 0.0f)
	{
		System->Tubes[x][y].DesX = System->Tubes[x][y].DesX + Buffer[x][y].DesX;
	}
	else
	{
		//printf("x: %d Y: %d %lf\n", x, y, System->Tubes[x][y].DesX + Buffer[x][y].DesX);
	}
	if(System->Tubes[x][y].DesY + Buffer[x][y].DesY <= (SIZE - 1) && System->Tubes[x][y].DesY + Buffer[x][y].DesY >= 0.0f)
	{
		System->Tubes[x][y].DesY = System->Tubes[x][y].DesY + Buffer[x][y].DesY;
	}
	else
	{
		//printf("x: %d Y: %d %lf\n", x, y, System->Tubes[x][y].DesY + Buffer[x][y].DesY);
	}
}