#include "hip/hip_runtime.h"
#include "main.cuh"

__global__
void applyTubes(FSystem* System, int Index, EReigonType** Reigons, FImage* Images, int* NumOfImages, int* ImageUsed)
{
	int x = blockIdx.x;
    int y = threadIdx.x;

    EReigonType TubeType = Reigons[x][y];

    //Check If Tube Is Image REigon
    if(TubeType == EReigonType::Input)
    {
        //Load In Image Data
        System[Index].Tubes[x][y].SrcX = Images[*ImageUsed].Pixels[x][y].SrcX * (SIZE - 1);
        System[Index].Tubes[x][y].SrcY = Images[*ImageUsed].Pixels[x][y].SrcY * (SIZE - 1);
        System[Index].Tubes[x][y].DesX = Images[*ImageUsed].Pixels[x][y].DesX * (SIZE - 1);
        System[Index].Tubes[x][y].DesY = Images[*ImageUsed].Pixels[x][y].DesY * (SIZE - 1);
    }
    else
    {
        //Load In Standard Chem Level
        System[Index].Tubes[x][y].SrcX = SIZE / 2;
        System[Index].Tubes[x][y].SrcY = SIZE / 2;
        System[Index].Tubes[x][y].DesX = SIZE / 2;
        System[Index].Tubes[x][y].DesY = SIZE / 2;
    }
}