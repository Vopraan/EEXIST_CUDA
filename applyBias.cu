#include "hip/hip_runtime.h"
#include "main.cuh"

__global__
void applyBias(FSystem* System, FBias** LatestBias, double* Mutation, int gen)
{
	int x = blockIdx.x;
	int y = threadIdx.x; 

	int RandomNum = fabsf((clock64() + 1) * (x + 23) * (y + 56) % 100000);

	if(RandomNum < *Mutation * 100000 || gen == 1)
	{
		//Create 4 Dffrent Random Values
		int RandomBias1 = fabsf((clock64() * (x + 1) * (y + 1)) % SIZE);
		int RandomBias2 = fabsf((clock64() * (x + 1 + clock64()) * (y + 1)) % SIZE);
		int RandomBias3 = fabsf((clock64() * (x + 3 * x) * (y + 1) * clock64()) % SIZE);
		int RandomBias4 = fabsf((clock64() * (x + y) * (y + -x + clock64() * clock64())) % SIZE);

		//Copy Bias Perfectly No Mutation
		System->Bias[x][y].SrcX = RandomBias1 - x;
		System->Bias[x][y].SrcY = RandomBias2 - y;
		System->Bias[x][y].DesX = RandomBias3 - x;
		System->Bias[x][y].DesY = RandomBias4 - y;
	}
	else
	{
		//Copy Bias Perfectly No Mutation
		System->Bias[x][y].SrcX = LatestBias[x][y].SrcX;
		System->Bias[x][y].SrcY = LatestBias[x][y].SrcY;
		System->Bias[x][y].DesX = LatestBias[x][y].DesX;
		System->Bias[x][y].DesY = LatestBias[x][y].DesY;
	}
}