#include "main.cuh"

void runTrainer(int* Generations, int* SystemsToRun, int* GuessesPerGen, int* UpdatesBeforeGuess, double* Karma, double* Mutation, int* Breeders, int* NumOfImages, FImage* Images, EReigonType** Reigons)
{
	//Reserve Sapce For Systems
	FSystem* Systems;
	hipMallocManaged(&Systems, sizeof(FSystem) * (*SystemsToRun));

	//Reserve Sapce For Bias
	FBias** Bias;
	hipMallocManaged(&Bias, sizeof(FBias*) * SIZE);
	for(int i = 0; i < SIZE; i++)
	{
		hipMallocManaged(&Bias[i], sizeof(FBias) * SIZE);
	}

	//Reserve Sapce For Buffer
	FTube** Buffer;
	hipMallocManaged(&Buffer, sizeof(FTube*) * SIZE);
	for(int i = 0; i < SIZE; i++)
	{
		hipMallocManaged(&Buffer[i], sizeof(FTube) * SIZE);
	}

	int* ImageUsed;
	hipMallocManaged(&ImageUsed, sizeof(int));

	int* TopPerformers;
	hipMallocManaged(&TopPerformers, *SystemsToRun * sizeof(int));


	//Iterate Through Gens
	for (int Gen = 1; Gen <= *Generations; Gen++)
	{
		printf("Gen: %d\n\n", Gen);

		for(int SysIndex = 0; SysIndex < *SystemsToRun; SysIndex++)
		{
			printf("SYS: %d\n", SysIndex);

			int test = rand() % 512;

			//Apply Bias
			applyBias <<< SIZE, SIZE >>> (&Systems[SysIndex], Bias, Mutation, Gen);
			hipDeviceSynchronize();

			for(int Tests = 0; Tests < *GuessesPerGen; Tests++)
			{
				*ImageUsed = 0;

				applyTubes <<< SIZE, SIZE >>> (Systems, SysIndex, Reigons, Images, NumOfImages, ImageUsed);
				hipDeviceSynchronize();

				//Iterate Though work updates
				for (int Update = 1; Update <= *UpdatesBeforeGuess; Update++)
				{
					//Clear Buffer
					clearBuffer <<< SIZE, SIZE >>> (Buffer);
					hipDeviceSynchronize();

					//Update System 
					updateSystem <<< SIZE, SIZE >>> (&Systems[SysIndex], Karma, Buffer, 10);
					hipDeviceSynchronize();

					
				}
			}

			//Score This Systems Bias
			scoreSystem(&Systems[SysIndex], Reigons, &Images[*ImageUsed]);
		}

		//Take Top Bias And Breed Them To Produce One New Bias Without Mutation
		breedBiases <<< SIZE, SIZE >>> (Systems, SystemsToRun, Bias, Breeders, TopPerformers);
		hipDeviceSynchronize();

		if((Gen - 1) % 10 == 0)
		{
			//char end[100];
			//sprintf(end, "%d", Gen);

			//char out[100] = "output";

			//strcat(out, end);
			//FILE* file = fopen(out, "w");
			//if(file != NULL)
			//{
				//output(file, Systems, SystemsToRun, Gen, *GuessesPerGen);
			//}
			//fclose(file);
		}
	}
}

/*
					for(int y = 0; y < 10; y++)
					{
						for(int x = 0; x < 10; x++)
						{
							printf("%3.0lf %3.0lf %3.0lf %3.0lf  ", Systems[SysIndex].Tubes[x][y].SrcX, Systems[SysIndex].Tubes[x][y].SrcY, Systems[SysIndex].Tubes[x][y].DesX, Systems[SysIndex].Tubes[x][y].DesY);
			
						}
						printf("\n");
					}
					printf("\n");

#include "main.cuh"

void runTrainer(int* Generations, int* SystemsToRun, int* GuessesPerGen, int* UpdatesBeforeGuess, double* Karma, double* Mutation, int* Breeders, int* NumOfImages, FImage* Images, EReigonType** Reigons)
{
	//Reserve Sapce For Systems
	FSystem* Systems;
	hipMallocManaged(&Systems, sizeof(FSystem) * (*SystemsToRun));

	//Reserve Sapce For Bias
	FBias** Bias;
	hipMallocManaged(&Bias, sizeof(FBias*) * SIZE);
	for(int i = 0; i < SIZE; i++)
	{
		hipMallocManaged(&Bias[i], sizeof(FBias) * SIZE);
	}

	//Reserve Sapce For Buffer
	FTube** Buffer;
	hipMallocManaged(&Buffer, sizeof(FTube*) * SIZE);
	for(int i = 0; i < SIZE; i++)
	{
		hipMallocManaged(&Buffer[i], sizeof(FTube) * SIZE);
	}

	int* ImageUsed;
	hipMallocManaged(&ImageUsed, sizeof(int));
	int* TopPerformers;
	hipMallocManaged(&TopPerformers, *SystemsToRun * sizeof(int));

	//Iterate Through Gens
	for (int Gen = 1; Gen <= *Generations; Gen++)
	{
		// Error code to check return values for CUDA calls
		hipError_t err = hipSuccess;

		printf("Gen: %d\n\n", Gen);

		//Iterate Through Systems Testing
		for (int System = 0; System < *SystemsToRun; System++)
		{
			printf("    System: %d\n", System);

			Systems[System].CorrectGuesses = 0;

			//Apply Bias
			//applyBias <<< SIZE, SIZE >>> (&Systems[System], Bias, Mutation, Gen);
			//hipDeviceSynchronize();
			err = hipGetLastError();

			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch Apply Bias kernel (error code %s)!\n", hipGetErrorString(err));
				//exit(EXIT_FAILURE);
			}

			//Iterate Through Number Of Times To Test Bias
			for (int Tests = 1; Tests <= *GuessesPerGen; Tests++)
			{
				//Apply Tubes With RAndomly Selected Image
				applyTubes <<< SIZE, SIZE >>> (Systems, System, Reigons, Images, NumOfImages, ImageUsed);
				hipDeviceSynchronize();
				err = hipGetLastError();

				*ImageUsed = rand() % *NumOfImages;

				if (err != hipSuccess)
				{
					fprintf(stderr, "Failed to launch Apply Tubes kernel (error code %s)!\n", hipGetErrorString(err));
					//exit(EXIT_FAILURE);
				}

				//Iterate Though work updates
				for (int Update = 1; Update <= *UpdatesBeforeGuess; Update++)
				{
					//Clear Buffer
					clearBuffer <<< SIZE, SIZE >>> (Buffer);
					hipDeviceSynchronize();
					err = hipGetLastError();

					if (err != hipSuccess)
					{
						fprintf(stderr, "Failed to launch Clear kernel (error code %s)!\n", hipGetErrorString(err));
						//exit(EXIT_FAILURE);
					}

					printf("Bias\n");
					for(int y = 0; y < 10; y++)
					{
						for(int x = 0; x < 10; x++)
						{
							printf("%3d %3d %3d %3d  ", Bias[x][y].SrcX, Bias[x][y].SrcY, Bias[x][y].DesX, Bias[x][y].DesY);
						}
						printf("\n");
					}

					printf("Tubes\n   ");
					for(int y = 0; y < 10; y++)
					{
						for(int x = 0; x < 10; x++)
						{
							printf("%3.0lf %3.0lf %3.0lf %3.0lf  ", Systems[System].Tubes[x][y].SrcX, Systems[System].Tubes[x][y].SrcY, Systems[System].Tubes[x][y].DesX, Systems[System].Tubes[x][y].DesY);
						}
						printf("\n   ");
					}

					printf("Buffer\n      ");
					for(int y = 0; y < 10; y++)
					{
						for(int x = 0; x < 10; x++)
						{
							printf("%3.0lf %3.0lf %3.0lf %3.0lf  ", Buffer[x][y].SrcX, Buffer[x][y].SrcY, Buffer[x][y].DesX, Buffer[x][y].DesY);
						}
						printf("\n      ");
					}
				
					//Update System 
					updateSystem <<< SIZE, SIZE >>> (&Systems[System], Karma, Buffer, 10);
					hipDeviceSynchronize();
					err = hipGetLastError();

					if (err != hipSuccess)
					{
						fprintf(stderr, "Failed to launch Update kernel (error code %s)!\n", hipGetErrorString(err));
						//exit(EXIT_FAILURE);
					}
				
					//Apply Buffer
					applyBuffer <<< SIZE, SIZE >>> (&Systems[System], Buffer);
					hipDeviceSynchronize();
					err = hipGetLastError();

					if (err != hipSuccess)
					{
						fprintf(stderr, "Failed to launch Apply kernel (error code %s)!\n", hipGetErrorString(err));
						//exit(EXIT_FAILURE);
					}
				}

				//Score This Systems Bias
				scoreSystem(&Systems[System], Reigons, &Images[*ImageUsed]);
			}
		}

		//Take Top Bias And Breed Them To Produce One New Bias Without Mutation
		//breedBiases <<< SIZE, SIZE >>> (Systems, SystemsToRun, Bias, Breeders, TopPerformers);
		//hipDeviceSynchronize();


	}
}
*/