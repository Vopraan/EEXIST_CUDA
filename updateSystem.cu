#include "hip/hip_runtime.h"
#include "main.cuh"

__global__ 
void updateSystem(FSystem* System, double* Karma, FTube** Buffer, double delta)
{
	int KarmIn = *Karma;

	int x = blockIdx.x;
	int y = threadIdx.x;

	//Where To Take Cehms From SRC
	int SrcX = (int)System->Tubes[x][y].SrcX;
    int SrcY = (int)System->Tubes[x][y].SrcY;

	//Where To Take Chems From Des
	int DesX = (int)System->Tubes[x][y].DesX;
	int DesY = (int)System->Tubes[x][y].DesY;

	//Add Offset For Src (Bias)
	SrcX = SrcX + System->Bias[x][y].SrcX;
	SrcY = SrcY + System->Bias[x][y].SrcY;

	//Add Offset For Des (Bias)
	DesX = DesX + System->Bias[x][y].DesX;
	DesY = DesY + System->Bias[x][y].DesY;

	//Iterate Through Potential Karma Tubes in X
	for (int Kx = -KarmIn; Kx <= KarmIn; Kx++)
	{
		//Iterate Through Potential Karma Tubes in Y
		for (int Ky = -KarmIn; Ky <= KarmIn; Ky++)
		{
			//Calcaulte Distance From ORgin Tube
			double KarmaDistance = sqrtf(powf(Kx, 2) + powf(Ky, 2));
			if (KarmaDistance <= *Karma)
			{
				//Figure Out What Tubes Take And Put Into
				int KSrcX = (SrcX + Kx + SIZE) % SIZE;
				int KSrcY = (SrcY + Ky + SIZE) % SIZE;
				int KDesX = (DesX + Kx + SIZE) % SIZE;
				int KDesY = (DesY + Ky + SIZE) % SIZE;

				//Caclaute Multiplier offset of Chemcail TRanfer (Based On How Far FRom Center KARMA Tube You are)
				double KarmaMultiplier = KarmaDistance / *Karma;

				//Perform SrcX Transfer
				if(System->Tubes[KSrcX][KSrcY].SrcX > (delta * KarmaMultiplier) && System->Tubes[KDesX][KDesY].SrcX < ((SIZE - 1) - (delta * KarmaMultiplier)))
				{
					System->Tubes[KSrcX][KSrcY].SrcX = System->Tubes[KSrcX][KSrcY].SrcX - (delta * KarmaMultiplier);
					System->Tubes[KDesX][KDesY].SrcX = System->Tubes[KDesX][KDesY].SrcX + (delta * KarmaMultiplier);
				}

				//Perform SrcY Transfer
				if(System->Tubes[KSrcX][KSrcY].SrcY > (delta * KarmaMultiplier) && System->Tubes[KDesX][KDesY].SrcY < ((SIZE - 1) - (delta * KarmaMultiplier)))
				{
					System->Tubes[KSrcX][KSrcY].SrcY = System->Tubes[KSrcX][KSrcY].SrcY - (delta * KarmaMultiplier);
					System->Tubes[KDesX][KDesY].SrcY = System->Tubes[KDesX][KDesY].SrcY + (delta * KarmaMultiplier);
				}

				//Perform DesX Transfer
				if(System->Tubes[KSrcX][KSrcY].DesX > (delta * KarmaMultiplier) && System->Tubes[KDesX][KDesY].DesX < ((SIZE - 1) - (delta * KarmaMultiplier)))
				{
					System->Tubes[KSrcX][KSrcY].DesX = System->Tubes[KSrcX][KSrcY].DesX - (delta * KarmaMultiplier);
					System->Tubes[KDesX][KDesY].DesX = System->Tubes[KDesX][KDesY].DesX + (delta * KarmaMultiplier);
				}

				//Perform DesY Transfer
				if(System->Tubes[KSrcX][KSrcY].DesY > (delta * KarmaMultiplier) && System->Tubes[KDesX][KDesY].DesY < ((SIZE - 1) - (delta * KarmaMultiplier)))
				{
					System->Tubes[KSrcX][KSrcY].DesY = System->Tubes[KSrcX][KSrcY].DesY - (delta * KarmaMultiplier);
					System->Tubes[KDesX][KDesY].DesY = System->Tubes[KDesX][KDesY].DesY + (delta * KarmaMultiplier);
				}
			}
		}
	}
}
/*

//Calcaultes The Ammount Of Cehm That Should Be TAken From Tube Based On APreameters Provided.
__device__ 
void GetChemToTake(double Karma, double KarmaDistance, double CurrentChemLevel, double TargetChemLevel, double* TargetAmountToTake)
{
	double Targ = *TargetAmountToTake;

	//Basically a precentage based on how far away this point is from Center
	double KarmaMultiplier = KarmaDistance / Karma;

	//Scale The Amount Of Chem Reuqesting By The Karma Diustance.
	*TargetAmountToTake = *TargetAmountToTake * KarmaMultiplier;

	//Make Sure There Is Some Chemical To Take
	if (CurrentChemLevel < Targ)
	{
		*TargetAmountToTake = CurrentChemLevel;

		if(TargetChemLevel + *TargetAmountToTake > (SIZE - 1))
		{
			*TargetAmountToTake = (TargetChemLevel + *TargetAmountToTake) - (SIZE - 1);
		}

		return;
	}

	if(TargetChemLevel + *TargetAmountToTake > (SIZE - 1))
	{
		*TargetAmountToTake = (TargetChemLevel + *TargetAmountToTake) - (SIZE - 1);
	}

	return;
}


				//Calcaute How May Chemicals To Take From SOurce And Be TRansfered To Des
				GetChemToTake(*Karma, KarmaDistance, System->Tubes[KSrcX][KSrcY].SrcX + Buffer[KSrcX][KSrcY].SrcX, System->Tubes[KDesX][KDesY].SrcX + Buffer[KDesX][KDesY].SrcX, &Take);
				double KSrcXTake = Take;
                Take = delta;
                
				GetChemToTake(*Karma, KarmaDistance, System->Tubes[KSrcX][KSrcY].SrcY + Buffer[KSrcX][KSrcY].SrcY, System->Tubes[KDesX][KDesY].SrcY + Buffer[KDesX][KDesY].SrcY, &Take);
				double KSrcYTake = Take;
				Take = delta;

				GetChemToTake(*Karma, KarmaDistance, System->Tubes[KSrcX][KSrcY].DesX + Buffer[KSrcX][KSrcY].DesX, System->Tubes[KDesX][KDesY].DesX + Buffer[KDesX][KDesY].DesX, &Take);
				double KDesXTake = Take;
                Take = delta;
                
				GetChemToTake(*Karma, KarmaDistance, System->Tubes[KSrcX][KSrcY].DesY + Buffer[KSrcX][KSrcY].DesY, System->Tubes[KDesX][KDesY].DesY + Buffer[KDesX][KDesY].DesY, &Take);
				double KDesYTake = Take;
				Take = delta;
*/