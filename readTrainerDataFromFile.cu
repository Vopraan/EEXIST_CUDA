#include "main.cuh"

void readTrainerDataFromFile(FILE* File, int* Generations, int* SystemsToRun, int* GuessesPerGen, int* UpdatesBeforeGuess, double* Karma, double* Mutation, int* Breeders, int* NumOfImages, FImage** Images, EReigonType*** Reigons)
{
    //Get Number Of Generations
	char NumOfGenerations[10];
	fgets(NumOfGenerations, 10, File);
	sscanf(NumOfGenerations, "%d", Generations);

	//Get Systems To Run
	char Systems[10];
	fgets(Systems, 10, File);
    sscanf(Systems, "%d", SystemsToRun);
    
	//Get Amount Of Times To Trial Each Bias Each Genrations
	char NumOfTrials[10];
	fgets(NumOfTrials, 10, File);
	sscanf(NumOfTrials, "%d", GuessesPerGen);

	//Get Runs Before ASking For Guess
	char RunsBeforeAsking[10];
	fgets(RunsBeforeAsking, 10, File);
    sscanf(RunsBeforeAsking, "%d", UpdatesBeforeGuess);
    
    //Get Karma
	char KarmaGet[32];
	fgets(KarmaGet, 32, File);
    sscanf(KarmaGet, "%lf", Karma);
    
    //Get Mutation Rate
	char MutationRate[32];
	fgets(MutationRate, 32, File);
    sscanf(MutationRate, "%lf", Mutation);
    
    //Get Number Of Biases To Select For Breeding
	char NumOfBreeders[10];
	fgets(NumOfBreeders, 10, File);
	sscanf(NumOfBreeders, "%d", Breeders);

	//Get Number Of Images In File
	char NumOfImagesInFile[10];
	fgets(NumOfImagesInFile, 10, File);
	sscanf(NumOfImagesInFile, "%d", NumOfImages);
    
	//Reserve Space For Images
	hipMallocManaged(Images, sizeof(FImage) * (*NumOfImages));

    //Reserve Space For Tube Reigons
	hipMallocManaged(Reigons, sizeof(EReigonType*) * SIZE);
    for(int i = 0; i < SIZE; i++)
    {
        hipMallocManaged(&Reigons[0][i], sizeof(EReigonType) * SIZE);
	}

	//Get Base Tube Chem Levels (Iterate Through X Rows)
	for (int y = 0; y < SIZE; y++)
	{
		char TubeTypeRow[1302];

		fgets(TubeTypeRow, 1302, File);

        //Iterate Through Y Rows
		for (int x = 0; x < SIZE; x++)
		{
            //Get Tube Type In Form Of String For This XY
			char point[2];
			point[0] = TubeTypeRow[2 * x];
			point[1] = '\0';

            //Scan String For Integer
			int ReigonType = 0;
			sscanf(point, "%d", &ReigonType);

            //Store Tube Type
			Reigons[0][x][y] = (EReigonType)ReigonType;
		}
	}

    //Iterate Through Number Of Known Images
	for (int i = 0; i < *NumOfImages; i++)
	{
		//There Is Empty Space Here In File So We Need To REad To Next Line
		char Empty[10];
		fgets(Empty, 10, File);

		//Get Image Size
		char ImageSize[20];
		fgets(ImageSize, 20, File);
		sscanf(ImageSize, "%d %d", &Images[0][i].x, &Images[0][i].y);

		//Get Image Correct Answer
		char ImageAnswer[5];
		fgets(ImageAnswer, 20, File);
		sscanf(ImageAnswer, "%d", &Images[0][i].CorrectIdentification);

		//Iterate Through Y Lines Of Image
		for (int y = 0; y < Images[0][i].y; y++)
		{
			//Get All Pixel RGB Values From Horizontal Row
			char* RGBLine = (char*)malloc(37 * Images[0][i].x + 2);
			fgets(RGBLine, 37 * Images[0][i].x + 2, File);

			//Itearte Through All Picles In That Row
			for (int x = 0; x < Images[0][i].x; x++)
			{
				char PixelChar[37];

				//Get Pixel Data And Put It Into Its Own String
				for (int j = 0; j < 37; j++)
				{
					PixelChar[j] = RGBLine[((x) * 37) + j];
				}

				//Scan Pixerl For RGB Information
				sscanf(PixelChar, "%lf %lf %lf %lf", &Images[0][i].Pixels[x][y].SrcX, &Images[0][i].Pixels[x][y].SrcY, &Images[0][i].Pixels[x][y].DesX, &Images[0][i].Pixels[x][y].DesY);
			}
		}
	}
}