#include "hip/hip_runtime.h"
#include "main.cuh"

__global__
void breedBiases(FSystem* Systems, int* NumOfSystems, FBias** OutBias, int* Breeders, int* TopPerformers)
{
	int x = blockIdx.x;
    int y = threadIdx.x;

    for(int i = 0; i < *NumOfSystems; i++)
    {
        for(int j = 0; j < *NumOfSystems; j++)
        {
            if(Systems[j].CorrectGuesses > Systems[i].CorrectGuesses)
            {
                TopPerformers[i] = j;
            }
        }
    }

    double SrcX = 0;
    double SrcY = 0;
    double DesX = 0;
    double DesY = 0;

    int count = 0;
    for(int i = 0; i < *Breeders; i++)
    {
        if(TopPerformers[i] >= 0)
        {
            SrcX = SrcX + Systems[TopPerformers[i]].Bias[x][y].SrcX;
            SrcY = SrcY + Systems[TopPerformers[i]].Bias[x][y].SrcY;
            DesX = DesX + Systems[TopPerformers[i]].Bias[x][y].DesX;
            DesY = DesY + Systems[TopPerformers[i]].Bias[x][y].DesY;

            count++;
        }
    }

    SrcX = SrcX / count;
    SrcY = SrcX / count;
    DesX = DesX / count;
    DesY = DesY / count;

    OutBias[x][y].SrcX = SrcX;
    OutBias[x][y].SrcX = SrcY;
    OutBias[x][y].SrcX = DesX;
    OutBias[x][y].SrcX = DesY;

}