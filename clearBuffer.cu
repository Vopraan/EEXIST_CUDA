#include "hip/hip_runtime.h"
#include "main.cuh"

__global__
void clearBuffer(FTube** Buffer)
{
	int x = blockIdx.x;
    int y = threadIdx.x;

    Buffer[x][y].SrcX = 0;
    Buffer[x][y].SrcY = 0;
    Buffer[x][y].DesX = 0;
    Buffer[x][y].DesY = 0;
}