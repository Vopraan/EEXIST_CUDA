#include "main.cuh"

int main()
{
	printf("This Program Runs EEXIST and trains a bias on a trainer data set provided.\n\n\n");

	//Training Variables
	int* Generations;			//How Many Gnenerations To Do Do
	int* SystemsToRun;			//How Many Systems To Train
	int* GuessesPerGen;			//How Many Times To Let Exist TRy To Guess Image EAch Generation
	int* UpdatesBeforeGuess;	//How Many Times To Update Each System Before Getting Answer From EEXIST
	double* Karma;				//How Many Times To Let Exist TRy To Guess Image EAch Generation
	double* Mutation;			//What Percentage Of Tubes To Make RAndom Vlaue For When Breeding Biases
	int* Breeders;				//How Many
	int* NumOfImages;			//How Mnay Images We Are TRaining Off of
	FImage* Images;				//Image Data
	EReigonType** Reigons;		//Reigon Data (Where To Place Images And Where To Read Output From)

	//Reserve Space On GPU For Training Data (Single VArs Only) (Arrays Will Be Reserved In Future Functions)
	hipMallocManaged(&Generations, sizeof(int*));
	hipMallocManaged(&SystemsToRun, sizeof(int*));
	hipMallocManaged(&GuessesPerGen, sizeof(int*));
	hipMallocManaged(&UpdatesBeforeGuess, sizeof(int*));
	hipMallocManaged(&Karma, sizeof(double*));
	hipMallocManaged(&Mutation, sizeof(double*));
	hipMallocManaged(&Breeders, sizeof(int*));
	hipMallocManaged(&NumOfImages, sizeof(int*));

	//Get The TRainer File That Has Data On How To Train
	FILE* Trainer = getTrainerFile();

	//Read InData From That File And Put It Into Our Varaibles
	readTrainerDataFromFile(Trainer, Generations, SystemsToRun, GuessesPerGen, UpdatesBeforeGuess, Karma, Mutation, Breeders, NumOfImages, &Images, &Reigons);

	//Train The AI
	runTrainer(Generations, SystemsToRun, GuessesPerGen, UpdatesBeforeGuess, Karma, Mutation, Breeders, NumOfImages, Images, Reigons);
}